#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
__device__ int d_size;

__global__ void partition (int *arr, int *arr_l, int *arr_h,long int n)
{
    int z = blockIdx.x*blockDim.x+threadIdx.x;
    d_size = 0;
    __syncthreads();
    if (z<n)
      {
        int h = arr_h[z];
        int l = arr_l[z];
        int x = arr[h];
        int i = (l - 1);
        int temp;
        for (int j = l; j <= h- 1; j++)
          {
            if (arr[j] <= x)
              {
                i++;
                temp = arr[i];
                arr[i] = arr[j];
                arr[j] = temp;
              }
          }
        temp = arr[i+1];
        arr[i+1] = arr[h];
        arr[h] = temp;
        int p = (i + 1);
        if (p-1 > l)
          {
            int ind = atomicAdd(&d_size, 1);
            arr_l[ind] = l;
            arr_h[ind] = p-1;  
          }
        if ( p+1 < h )
          {
            int ind = atomicAdd(&d_size, 1);
            arr_l[ind] = p+1;
            arr_h[ind] = h; 
          }
      }
}
 
void quickSortIterative (int arr[],long int l,long int h)
{
    int lstack[ h - l + 1 ], hstack[ h - l + 1];
 
    int *d_d, *d_l, *d_h;
    long int top = -1;
 
    lstack[ ++top ] = l;
    hstack[ top ] = h;

    hipMalloc(&d_d, (h-l+1)*sizeof(int));
    hipMemcpy(d_d, arr,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc(&d_l, (h-l+1)*sizeof(int));
    hipMemcpy(d_l, lstack,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc(&d_h, (h-l+1)*sizeof(int));
    hipMemcpy(d_h, hstack,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);
    int n_t = 1;
    int n_b = 1;
    long int n_i = 1; 
    while ( n_i > 0 )
    {
        partition<<<n_b,n_t>>>( d_d, d_l, d_h, n_i);
        int answer;
        hipMemcpyFromSymbol(&answer, HIP_SYMBOL(d_size), sizeof(int), 0, hipMemcpyDeviceToHost); 
        if (answer < 1024)
          {
            n_t = answer;
          }
        else
          {
            n_t = 1024;
            n_b = answer/n_t + (answer%n_t==0?0:1);
          }
        n_i = answer;
        hipMemcpy(arr, d_d,(h-l+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
}
 

 
int main()
{
    long int n=1024*1;
    int arr[n];
    srand(time(NULL));
    for (int i = 0; i<n; i++)
       {
         arr[i] = rand ()%10000;
       }
    n = sizeof( arr ) / sizeof( *arr );
    hipEvent_t start,end;
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start);
    quickSortIterative( arr, 0, n - 1 );
    hipEventRecord(end);
		hipEventSynchronize(end);
		float milliseconds=0;
		hipEventElapsedTime(&milliseconds,start,end);
		double timeTaken=(double)milliseconds;
    double throughput = (n*sizeof(int))/(timeTaken);
    printf("%f,%f",timeTaken/1000,throughput);
    return 0;
}